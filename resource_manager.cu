#include "resource_manager.h"
#include <hip/hip_runtime.h>

ResourceManager::ResourceManager() : size((M + 2) * (N + 2) * (O + 2)) {
  hipMalloc(&d_dev, size * sizeof(float));
  hipMalloc(&d0_dev, size * sizeof(float));
  hipMalloc(&u_dev, size * sizeof(float));
  hipMalloc(&u0_dev, size * sizeof(float));
  hipMalloc(&v_dev, size * sizeof(float));
  hipMalloc(&v0_dev, size * sizeof(float));
  hipMalloc(&w_dev, size * sizeof(float));
  hipMalloc(&w0_dev, size * sizeof(float));
  hipMalloc(&max_change, sizeof(float));

  hipMemset(d_dev, 0, size * sizeof(float));
  hipMemset(d0_dev, 0, size * sizeof(float));
  hipMemset(u_dev, 0, size * sizeof(float));
  hipMemset(u0_dev, 0, size * sizeof(float));
  hipMemset(v_dev, 0, size * sizeof(float));
  hipMemset(v0_dev, 0, size * sizeof(float));
  hipMemset(w_dev, 0, size * sizeof(float));
  hipMemset(w0_dev, 0, size * sizeof(float));
}

ResourceManager::~ResourceManager() {
  hipFree(d_dev);
  hipFree(d0_dev);
  hipFree(u_dev);
  hipFree(u0_dev);
  hipFree(v_dev);
  hipFree(v0_dev);
  hipFree(w_dev);
  hipFree(w0_dev);
  hipFree(max_change);
}
