#include "resource_manager.h"
#include <hip/hip_runtime.h>

ResourceManager::ResourceManager() : size((M + 2) * (N + 2) * (O + 2)) {
  hipMalloc(&x_dev, size * sizeof(float));
  hipMalloc(&x0_dev, size * sizeof(float));
  hipMalloc(&d_dev, size * sizeof(float));
  hipMalloc(&d0_dev, size * sizeof(float));
  hipMalloc(&u_dev, size * sizeof(float));
  hipMalloc(&v_dev, size * sizeof(float));
  hipMalloc(&w_dev, size * sizeof(float));
}

ResourceManager::~ResourceManager() {
  hipFree(x_dev);
  hipFree(x0_dev);
}
